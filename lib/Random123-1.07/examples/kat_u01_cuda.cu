#include "hip/hip_runtime.h"
/*
Copyright 2010-2011, D. E. Shaw Research.
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright
  notice, this list of conditions, and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions, and the following disclaimer in the
  documentation and/or other materials provided with the distribution.

* Neither the name of D. E. Shaw Research nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#include "kat_u01_main.h"
#include "util_cuda.h"

#define KAT_KERNEL __global__
#define KAT_THREADID (blockDim.x * blockIdx.x + threadIdx.x)

#include "kat_u01_dev_execute.h"

void host_execute_tests(uint64_t *tests, size_t ntests, KatU01Result *results){
    CUDAInfo *infop;
    KatU01Result *results_dev;
    uint64_t *tests_dev;
    size_t tests_sz, results_sz;

    infop = cuda_init(NULL);
    tests_sz = ntests * sizeof(*tests_dev);
    CHECKCALL(hipMalloc(&tests_dev, tests_sz));
    CHECKCALL(hipMemcpy(tests_dev, tests, tests_sz, hipMemcpyHostToDevice));
    
    results_sz = ntests * sizeof(*results);
    CHECKCALL(hipMalloc(&results_dev, results_sz));

    printf("starting on %lu blocks with 1 threads/block\n", (unsigned long)ntests);
    fflush(stdout);

    dev_execute_tests<<<ntests, 1>>>(tests_dev, results_dev);

    CHECKCALL(hipDeviceSynchronize());
    CHECKCALL(hipMemcpy(results, results_dev, results_sz, hipMemcpyDeviceToHost));
    CHECKCALL(hipFree(tests_dev));
    CHECKCALL(hipFree(results_dev));
    cuda_done(infop);
}

